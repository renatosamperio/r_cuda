#include <cmath>
#include <cuda_rand.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

void testMalloc()
{
  size_t
    fbytes = sizeof(double);
  double
    *d_mat;
    
    hipMalloc((void**)&d_mat, 200*50*fbytes);
    hipFree(d_mat);
}